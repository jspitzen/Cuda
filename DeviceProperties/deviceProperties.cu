#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <math.h>


int main() {
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device number: %d\n", i);
		printf("  Device name: %s\n",prop.name);

	}
}